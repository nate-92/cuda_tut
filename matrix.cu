#include "hip/hip_runtime.h"
#include "matrix.h"
#include <stdio.h>
#include <iostream>

__global__ void add(Matrix * m1, Matrix * m2){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    m1->matrix[index] += m2->matrix[index];
}

__global__ void displaySize(Matrix * m1){
    printf("Rows: %d \t Cols: %d\n",m1->rows, m1->cols);
}

__global__ void displayMatrix(Matrix * m){
    for(int i = 0; i < m->rows; i++){
        for(int j = 0; j < m->cols; j++){
            printf("%f\t",m->matrix[i * m->cols + j]);
        }
        printf("\n");
    }
}

void display(Matrix * m){
    for(int i = 0; i < m->rows; i++){
        for(int j = 0; j < m->cols; j++){
            std::cout<< m->matrix[i * m->cols + j] << "\t";
        }
        std::cout << std::endl;
    }
}

void sync(){
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        fprintf(stderr,"Error: %s\n",hipGetErrorString(error));
        exit(1);
    }
}

int main(){
    Matrix * h_m1 = makeMatrix(5,5,1), * h_m2 = makeMatrix(5,5,1);
    std::cout << "M1\n";
    display(h_m1);
    std::cout << std::endl;
    std::cout << "M2\n";
    display(h_m2);
    std::cout << std::endl;

    

    Matrix * d_m1 = makeDeviceMatrix(h_m1);
    Matrix *d_m2 = makeDeviceMatrix(h_m2);

    std::cout << "M1 Size\n";
    displaySize<<<1,1>>>(d_m1);
    sync();

    std::cout << "M2 Size\n";
    displaySize<<<1,1>>>(d_m2);
    sync();

    add<<<h_m1->rows,h_m1->cols>>>(d_m1, d_m2);
    sync();

    std::cout << "Addition\n";
    displayMatrix<<<1,1>>>(d_m1);
    sync();

    /*free(h_m1);
    h_m1 = copyMatrixToHost(d_m1);
    display(h_m1);*/


}