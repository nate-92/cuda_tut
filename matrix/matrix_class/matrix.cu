#include "hip/hip_runtime.h"
#include <iostream>
#include "matrix.cuh"

__global__ void add(Matrix * m1, Matrix * m2){
    int index = threadIdx.x * m1->getCols() + blockIdx.x;
    m1->add(index,m2->getVal(index));
}

void sync(){
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        fprintf(stderr,"Error: %s\n",hipGetErrorString(error));
        exit(1);
    }
}
Matrix * moveMatrixToDevice(Matrix * h_m);

int main(){
    Matrix * h_m1 = new Matrix(8,8,1), * h_m2 = new Matrix(8,8,1);

    std::cout << "M1\n";
    h_m1->display();
    std::cout << "M2\n";
    h_m2->display();

    h_m1->matToDevice();
    h_m2->matToDevice();


    Matrix * d_m1 = moveMatrixToDevice(h_m1), * d_m2 = moveMatrixToDevice(h_m2);

    std::cout << "Addition\n";
    add<<<8,8>>>(d_m1,d_m2);
    sync();

    std::cout << "After Addition\n";
    std::cout << "M1\n";
    h_m1->matToHost();
    h_m1->display();
    std::cout << "M2\n";
    h_m1->matToHost();
    h_m2->display();
}

Matrix * moveMatrixToDevice(Matrix * h_m){
    Matrix * d_m;
    //Allocate Space
    hipMalloc((void **) &d_m,sizeof(Matrix));
    //Copy to device
    hipMemcpy(d_m, h_m, sizeof(Matrix),hipMemcpyHostToDevice);
    return d_m;
}