#include "set.cuh"
#define DEBUG


int main(){
    std::vector<int> set(698);
    for(int i = 0; i < set.size(); i++){
        set[i] = i+1;
    }


    Set<int> * h_s = new Set<int>(set,2);
    Set<int> * d_s = setToGPU(h_s);

    #ifdef DEBUG
        std::cout << "Locating Cofaces\n";
        std::cout << "Local Vec Size: " << h_s->getVecSize() << "\tLocal Dim: " << h_s->getDim() << std::endl;
        display<<<1,1>>>(d_s);
    #endif

    sync();

    locateCofaces<<<1,h_s->getVecSize()>>>(d_s);
    sync();

    #ifdef DEBUG
        std::cout << "Finished locating cofaces\n";
    #endif

    h_s = setToHost(d_s);
    h_s->display();


    std::vector<int> * cofaces = h_s->getCofaces(set);
    /*int size = h_s->getCofaceSize();

    for(int i = 0; i < size; i++){
        std::vector<int> coface = cofaces[i];
        for(int j = 0; j < coface.size(); j++){
            std::cout << coface[j];
        }
        std::cout << std::endl;
    }*/
    //s.makeCofaceSet(2);
}